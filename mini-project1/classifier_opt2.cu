#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define TILE_NI 256

bool verify(const float *ref, const half *gpu_half, size_t N, float tolerance = 0.5) {
    float *gpu_float = (float *)malloc(N * sizeof(float));
    for (size_t i = 0; i < N; ++i) {
        gpu_float[i] = __half2float(gpu_half[i]);
    }

    size_t errors = 0;
    for (size_t i = 0; i < N; ++i) {
        float diff = fabsf(ref[i] - gpu_float[i]);
        bool error_cond = diff > tolerance && diff > fabsf(ref[i] * tolerance);
        if (ref[i] == 0.0f)
            error_cond = diff > tolerance;

        if (error_cond) {
            if (errors < 10) {
                fprintf(stderr, " Verification failed at index %zu: Ref=%.6f, GPU(FP16)=%.6f, Diff=%.6f\n", i, ref[i], gpu_float[i], diff);
            }
            errors++;
        }
    }
    if (errors == 0) {
        printf("Verification Successful!\n");
        return true;
    } else {
        printf("Verification FAILED with %zu errors!\n", errors);
        return false;
    }
}

void classifier_cpu(float *output, const float *input, const float *weights,
                    int B, int Ni, int Nn) {
    for (int b = 0; b < B; ++b) {
        for (int nn = 0; nn < Nn; ++nn) {
            float sum = 0.0f;
            for (int ni = 0; ni < Ni; ++ni) {
                size_t input_idx = (size_t)b * Ni + ni;
                size_t weight_idx = (size_t)nn * Ni + ni;  // Weights[Nn x Ni] layout
                sum += input[input_idx] * weights[weight_idx];
            }
            size_t output_idx = (size_t)b * Nn + nn;
            output[output_idx] = sum;
        }
    }
}

__global__ void classifier_kernel_fp16_shared_memory(
    half *output, const half *input, const half *weights,
    int B, int Ni, int Nn) {
    int nn = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.y;

    __shared__ half input_tile[TILE_NI];
    // __shared__ half weight_tile[TILE_NI];

    if (nn < Nn && b < B) {
        half sum_fp32 = 0.0f;

        for (int ni_base = 0; ni_base < Ni; ni_base += TILE_NI) {
            const int tile_idx = threadIdx.x;           // Thread index maps directly to index within the tile
            const int current_ni = ni_base + tile_idx;  // Global ni index for this thread's element

            // Load input element into shared memory (check bounds)
            if (current_ni < Ni) {
                const size_t input_idx_global = (size_t)b * Ni + current_ni;
                input_tile[tile_idx] = input[input_idx_global];
            } else {
                input_tile[tile_idx] = 0.0f;  // Pad with zero if out of bounds
            }

            // // Load weight element into shared memory (check bounds)
            // if (current_ni < Ni) {
            //     // Weight Index: [nn, current_ni]
            //     const size_t weight_idx_global = (size_t)nn * Ni + current_ni;
            //     weight_tile[tile_idx] = weights[weight_idx_global];
            // } else {
            //     weight_tile[tile_idx] = __float2half(0.0f);  // Pad with zero if out of bounds
            // }

            __syncthreads();

            const int current_tile_size = min(TILE_NI, Ni - ni_base);
            #pragma unroll
            for (int k = 0; k < current_tile_size; ++k) {
                const int weight_ni = ni_base + k;  // Global ni index for weight element
                const size_t weight_idx_global = (size_t)nn * Ni + weight_ni;
                // sum_fp32 += __half2float(input_tile[k]) * __half2float(weight_tile[k]);
                sum_fp32 += input_tile[k] * weights[weight_idx_global];
            }

            __syncthreads();
        }

        size_t output_idx = (size_t)b * Nn + nn;
        output[output_idx] = sum_fp32;
    }
}

// --- Main Verification Function ---
int main(int argc, char *argv[]) {
    // int Ni = 25088, Nn = 4096;
    // int B = 16;

    if (argc != 5) {
        fprintf(stderr, "Error: This program requires <Ni> <Nn> <B> <CPU verify> arguments.\n");
        return 1;
    }

    int Ni, Nn, B;

    Ni = atoi(argv[1]);
    Nn = atoi(argv[2]);
    B = atoi(argv[3]);
    bool CPU_verify = strcmp(argv[4], "true") == 0;

    printf("Ni = %d\n", Ni);
    printf("Nn = %d\n", Nn);
    printf("B  = %d\n", B);

    // Calculate sizes
    size_t input_size = (size_t)B * Ni;
    size_t weight_size = (size_t)Nn * Ni;  // Assuming Nn x Ni layout
    size_t output_size = (size_t)B * Nn;

    printf("Input: %d x %d, Weights: %d x %d, Output: %d x %d\n",
           B, Ni, Nn, Ni, B, Nn);
    printf("Total elements: Input= %zu, Weights=%zu, Output=%zu\n", input_size, weight_size, output_size);

    // Allocate Host Memory
    float *h_input_f = (float *)malloc(input_size * sizeof(float));
    float *h_weights_f = (float *)malloc(weight_size * sizeof(float));
    half *h_input_h = (half *)malloc(input_size * sizeof(half));
    half *h_weights_h = (half *)malloc(weight_size * sizeof(half));
    half *h_output_gpu_h = (half *)malloc(output_size * sizeof(half));
    float *h_output_cpu_f = (float *)malloc(output_size * sizeof(float));
    if (!h_input_f || !h_weights_f || !h_input_f || !h_weights_f || !h_output_gpu_h || !h_output_cpu_f) {
        fprintf(stderr, "Failed to allocate host memory!\n");
        return 1;
    }

    // Initialize Host Data (using fixed seed for reproducibility)
    srand(456);  // Different seed than conv2d
    for (size_t i = 0; i < input_size; ++i) {
        h_input_f[i] = (float)(rand() % 20 - 10) / 20.0f;  // Smaller range helps FP16
        h_input_h[i] = __float2half(h_input_f[i]);
    }
    for (size_t i = 0; i < weight_size; ++i) {
        h_weights_f[i] = (float)(rand() % 10 - 5) / 50.0f;  // Smaller weights/range
        h_weights_h[i] = __float2half(h_weights_f[i]);
    }

    // Allocate Device Memory
    half *d_input, *d_weights, *d_output;
    hipMalloc((void **)&d_input, input_size * sizeof(half));
    hipMalloc((void **)&d_weights, weight_size * sizeof(half));
    hipMalloc((void **)&d_output, output_size * sizeof(half));

    // Copy Data Host -> Device
    printf("Copying data to GPU...\n");
    hipMemcpy(d_input, h_input_h, input_size * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights_h, weight_size * sizeof(half), hipMemcpyHostToDevice);
    printf("Copying done.\n");

    // --- GPU Execution ---
    dim3 threadsPerBlock(TILE_NI);                                      // 1D block for classifier
    dim3 gridDim((Nn + threadsPerBlock.x - 1) / threadsPerBlock.x, B);  // Grid covers outputs and batch
    printf("Running Classifier GPU Kernel...\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    classifier_kernel_fp16_shared_memory<<<gridDim, threadsPerBlock>>>(
        d_output, d_input, d_weights, B, Ni, Nn);

    hipEventRecord(stop);

    hipGetLastError();
    hipDeviceSynchronize();
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Kernel finished in %.3f ms.\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("GPU Kernel finished.\n");

    // Copy Result Device -> Host
    printf("Copying result from GPU...\n");
    hipMemcpy(h_output_gpu_h, d_output, output_size * sizeof(half), hipMemcpyDeviceToHost);
    printf("Copying done.\n");

    if (CPU_verify) {
        // --- CPU Execution ---
        printf("Running Classifier CPU reference...\n");
        classifier_cpu(h_output_cpu_f, h_input_f, h_weights_f, B, Ni, Nn);
        printf("CPU reference finished.\n");

        // --- Verification ---
        printf("Verifying results...\n");
        verify(h_output_cpu_f, h_output_gpu_h, output_size);
    }

    // --- Cleanup ---
    printf("Cleaning up memory...\n");
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
    free(h_input_f);
    free(h_weights_f);
    free(h_input_h);
    free(h_weights_h);
    free(h_output_gpu_h);
    free(h_output_cpu_f);
    printf("Cleanup done.\n");

    return 0;
}